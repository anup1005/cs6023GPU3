#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;

__device__ int temp;
// __device__ int count;
ofstream outfile; // The handle for printing the output
/******************************Write your kerenels here ************************************/
__global__ void settemp(int lastind){
    temp=lastind;
}

// __global__ void setcount(int lastind){
//     count=0;
// }


__global__ void mykernel(int *d_apr,int total){
    int tid= blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<total){
        if(d_apr[tid]==0){
            atomicMax(&temp,tid);
        }
    }
}




//favkernel<<<numblocks,1024>>>(d_offset,d_csrList,startedge_ind,totaledges,temp,d_aid,mynode)
__global__ void favkernel(int* d_offset,int* d_csrList,int startind,int total,int *d_aid,int value){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<total){
        int csrind=startind+tid;
        if(value==1){
            atomicMax(&temp,d_csrList[csrind]);
            atomicAdd(&d_aid[d_csrList[csrind]],value);
        //atomicInc((unsigned int *)(d_aid[d_csrList[csrind]]),INT_MAX);}
        }
        else if(value==-1){
            atomicAdd(&d_aid[d_csrList[csrind]],value);
        }
}
}
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));




    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex,*h_aid;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    h_aid = (int*)malloc(V*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));
    memset(h_aid, 0, V*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
	hipMalloc(&d_activeVertex, L*sizeof(int));
    hipMemset(d_activeVertex,0,L*sizeof(int));
    hipMemset(d_aid,0,V*sizeof(int));


/***Important***/


// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/
// int *temp;
// hipMalloc(&temp,1*sizeof(int));
// hipMemset(&temp,69,1*sizeof(int));



// int a=69;
// settemp<<<1,1>>>(a);
// hipDeviceSynchronize();
// int target;
// hipMemcpyFromSymbol(&target,HIP_SYMBOL(temp), sizeof(int));
// printf("\ntemp in main is target and its value is %d\n",target);

int destination=-11;



int startind=0;
int lastind;
int total=min(10000,V);
int numblocks=  ceil(float(total)/ 1024);
int a=0;
settemp<<<1,1>>>(a);
hipDeviceSynchronize();
mykernel<<<numblocks,1024>>>(d_apr,total);

hipMemcpyFromSymbol(&lastind,HIP_SYMBOL(temp), sizeof(int));


printf("last node in the first layer is %d",lastind);



int* Vlayers=(int *)malloc(L*sizeof(int));
memset(Vlayers, 0, L*sizeof(int));
Vlayers[0]=lastind+1;
printf("no. of layers are %d\n",L);

// printf("\n\n\nprinting Vlayers before editing\n");
// for(int i=0;i<L;i++){
//     printf("%d   ",Vlayers[i]);
// }
printf("taarak mehta\n");


hipDeviceSynchronize();
int ch=0;

printf("INITIALLY startind is %d lastind is %d\n",startind,lastind);

while(lastind!=V-1){
    settemp<<<1,1>>>(lastind);
    for(int mynode=startind;mynode<=lastind;mynode++){

        int startedge_ind=h_offset[mynode];
        int endedge_ind=h_offset[mynode+1]-1;
        int totaledges= endedge_ind-startedge_ind+1;
        int numblocks=ceil(float(totaledges)/1024);
        favkernel<<<numblocks,1024>>>(d_offset,d_csrList,startedge_ind,totaledges,d_aid,1);

    }
    hipDeviceSynchronize();
    int target;
    hipMemcpyFromSymbol(&target,HIP_SYMBOL(temp), sizeof(int));
    startind=lastind+1;
    lastind=target;
    ch+=1;
    Vlayers[ch]=lastind+1;
}




printf("\n\n\n");
printf("hello MR. Anup");
/*
    mere Vlayers s 5,10,14,17,20 aa gaye h
*/

/*
    mere d_aid m sare edges k degree aa gaye h
*/

hipMemcpy(h_aid, d_aid, V*sizeof(int), hipMemcpyDeviceToHost);

// printf("\n\n\nprinting Vlayers after editing\n");
// for(int i=0;i<L;i++){
//     printf("%d   ",Vlayers[i]);
// }



// printf("\nprinting just indegree after accounting each and every edge\n");
// for(int i=0;i<V;i++){
//     printf("%d  ",h_aid[i]);
// }
printf("\n anup just final step remaining\n");



h_activeVertex[0]=Vlayers[0];

for(int i=1;i<L;i++){
    printf("layer no. %d",i);
    int startnode=Vlayers[i-1];
    int lastnode=Vlayers[i]-1;
    //printf("layer%d    startnode%d   lastnode%d   ",i,startnode,lastnode);
    int count=0;
    int lastinactive=INT_MIN;
    for(int mynode=startnode;mynode<=lastnode;mynode++){
        if(h_aid[mynode]>=h_apr[mynode]){
            //printf("node active %d\n",mynode);
            count++;
           // printf("+1 count =  %d",count );
        }
       else{
        if(lastinactive+2==mynode){

            int startedge_ind=h_offset[mynode-1];
            int endedge_ind=h_offset[mynode]-1;
            int totaledges= endedge_ind-startedge_ind+1;
            int numblocks=ceil(float(totaledges)/1024);
            count--;
            favkernel<<<numblocks,1024>>>(d_offset,d_csrList,startedge_ind,totaledges,d_aid,-1);
            hipMemcpy(h_aid, d_aid, V*sizeof(int), hipMemcpyDeviceToHost);
        }
        lastinactive=mynode;
        int startedge_ind=h_offset[mynode];
        int endedge_ind=h_offset[mynode+1]-1;
        int totaledges= endedge_ind-startedge_ind+1;
        int numblocks=ceil(float(totaledges)/1024);
        favkernel<<<numblocks,1024>>>(d_offset,d_csrList,startedge_ind,totaledges,d_aid,-1);
        hipMemcpy(h_aid, d_aid, V*sizeof(int), hipMemcpyDeviceToHost);
    }
}
    printf("   %d  \n",count);
    hipDeviceSynchronize();
    h_activeVertex[i]=count;
}
printf("\n my answeris\n");
for(int i=0;i<L;i++){
    printf("%d   ",h_activeVertex[i]);
}
printf("\nhello world\n");




    
 

    
   
    
    

     

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
